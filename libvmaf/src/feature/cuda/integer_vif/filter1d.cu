#include "hip/hip_runtime.h"
/**
 *
 *  Copyright 2016-2023 Netflix, Inc.
 *  Copyright 2021 NVIDIA Corporation.
 *
 *     Licensed under the BSD+Patent License (the "License");
 *     you may not use this file except in compliance with the License.
 *     You may obtain a copy of the License at
 *
 *         https://opensource.org/licenses/BSDplusPatent
 *
 *     Unless required by applicable law or agreed to in writing, software
 *     distributed under the License is distributed on an "AS IS" BASIS,
 *     WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *     See the License for the specific language governing permissions and
 *     limitations under the License.
 *
 */

#include "cuda_helper.cuh"
#include "cuda/integer_vif_cuda.h"

#include "common.h"

#include "vif_statistics.cuh"

template <typename alignment_type = uint2, int fwidth_0 = 17, int fwidth_1 = 9>
__device__ __forceinline__ void filter1d_8_vertical_kernel(VifBufferCuda buf, uint8_t* ref_in, uint8_t* dis_in,
        int w, int h, filter_table_stuct vif_filt_s0) {
    using writeback_type = uint4;
    constexpr int val_per_thread = sizeof(alignment_type);
    static_assert(val_per_thread % 4 == 0 && val_per_thread <= 16,
            "val per thread bust be divisible by 4 and under 16");
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int x_start = (blockIdx.x * blockDim.x + threadIdx.x) * val_per_thread;
    if (x_start < w && y < h) {
        const int stride_tmp = buf.stride_tmp / sizeof(uint32_t);
        __align__(sizeof(writeback_type)) uint32_t accum_mu1[val_per_thread] = {0};
        __align__(sizeof(writeback_type)) uint32_t accum_mu2[val_per_thread] = {0};
        __align__(sizeof(writeback_type)) uint32_t accum_ref[val_per_thread] = {0};
        __align__(sizeof(writeback_type)) uint32_t accum_dis[val_per_thread] = {0};
        __align__(sizeof(writeback_type)) uint32_t accum_ref_dis[val_per_thread] = {0};
        __align__(sizeof(writeback_type)) uint32_t accum_ref_rd[val_per_thread] = {0};
        __align__(sizeof(writeback_type)) uint32_t accum_dis_rd[val_per_thread] = {0};
        union {
            uint8_t ref[val_per_thread];
            alignment_type ref_aligned;
        };
        union {
            uint8_t dis[val_per_thread];
            alignment_type dis_aligned;
        };
        for (int fi = 0; fi < fwidth_0; ++fi) {
            const int ii = y - fwidth_0 / 2;
            unsigned int ii_check = abs(ii + fi);
            if (ii_check >= h) {
                ii_check = 2 * h - ii_check - 2;
            }
            ref_aligned = *reinterpret_cast<alignment_type *>(
                    &(ref_in[ii_check * buf.stride + x_start]));
            dis_aligned = *reinterpret_cast<alignment_type *>(
                    &(dis_in[ii_check * buf.stride + x_start]));
            for (int off = 0; off < val_per_thread; ++off) {
                const int j = x_start + off;
                if (j < w) {
                    const uint32_t fcoeff = vif_filt_s0.filter[0][fi];
                    const uint32_t ref_val = ref[off];
                    const uint32_t dis_val = dis[off];
                    const uint32_t img_coeff_ref = fcoeff * (uint32_t)ref_val;
                    const uint32_t img_coeff_dis = fcoeff * (uint32_t)dis_val;
                    accum_mu1[off] += img_coeff_ref;
                    accum_mu2[off] += img_coeff_dis;
                    accum_ref[off] += img_coeff_ref * (uint32_t)ref_val;
                    accum_dis[off] += img_coeff_dis * (uint32_t)dis_val;
                    accum_ref_dis[off] += img_coeff_ref * (uint32_t)dis_val;
                    if (fi >= (fwidth_0 - fwidth_1) / 2 &&
                            fi < (fwidth_0 - (fwidth_0 - fwidth_1) / 2)) {
                        const uint16_t fcoeff_rd =
                            vif_filt_s0.filter[1][fi - ((fwidth_0 - fwidth_1) / 2)];
                        accum_ref_rd[off] += fcoeff_rd * ref_val;
                        accum_dis_rd[off] += fcoeff_rd * dis_val;
                    }
                }
            }
        }
        for (int off = 0; off < val_per_thread; ++off) {
            accum_mu1[off] = (accum_mu1[off] + 128) >> 8;
            accum_mu2[off] = (accum_mu2[off] + 128) >> 8;
            accum_ref_rd[off] = (accum_ref_rd[off] + 128) >> 8;
            accum_dis_rd[off] = (accum_dis_rd[off] + 128) >> 8;
        }
        for (int idx = 0; idx < val_per_thread; idx += sizeof(writeback_type) / sizeof(uint32_t)) {
            const int buffer_idx = y * stride_tmp + x_start + idx;
            if (x_start + idx < w) {
                *reinterpret_cast<writeback_type *>(&buf.tmp.mu1[buffer_idx]) =
                    *reinterpret_cast<writeback_type *>(&accum_mu1[idx]);
                *reinterpret_cast<writeback_type *>(&buf.tmp.mu2[buffer_idx]) =
                    *reinterpret_cast<writeback_type *>(&accum_mu2[idx]);
                *reinterpret_cast<writeback_type *>(&buf.tmp.ref[buffer_idx]) =
                    *reinterpret_cast<writeback_type *>(&accum_ref[idx]);
                *reinterpret_cast<writeback_type *>(&buf.tmp.dis[buffer_idx]) =
                    *reinterpret_cast<writeback_type *>(&accum_dis[idx]);
                *reinterpret_cast<writeback_type *>(&buf.tmp.ref_dis[buffer_idx]) =
                    *reinterpret_cast<writeback_type *>(&accum_ref_dis[idx]);
                *reinterpret_cast<writeback_type *>(&buf.tmp.ref_convol[buffer_idx]) =
                    *reinterpret_cast<writeback_type *>(&accum_ref_rd[idx]);
                *reinterpret_cast<writeback_type *>(&buf.tmp.dis_convol[buffer_idx]) =
                    *reinterpret_cast<writeback_type *>(&accum_dis_rd[idx]);
            }
        }
    }
}

template <int val_per_thread = 1, int fwidth_0 = 17, int fwidth_1 = 9>
__device__ __forceinline__ void filter1d_8_horizontal_kernel(VifBufferCuda buf, int w, int h,
        filter_table_stuct vif_filt_s0,
        double vif_enhn_gain_limit,
        vif_accums *accum) {
    static_assert(val_per_thread % 2 == 0,
            "val_per_thread must be divisible by 2");
    int y = blockIdx.y;
    int x_start = (blockIdx.x * blockDim.x + threadIdx.x) * val_per_thread;
    if (y < h && x_start < w) {

        union {
            vif_accums thread_accum;
            int64_t thread_accum_i64[7] = {0};
        };

        uint32_t accum_mu1[val_per_thread] = {0};
        uint32_t accum_mu2[val_per_thread] = {0};
        uint32_t accum_ref[val_per_thread] = {0};
        uint32_t accum_dis[val_per_thread] = {0};
        uint32_t accum_ref_dis[val_per_thread] = {0};
        uint64_t accum_ref_tmp[val_per_thread] = {0};
        uint64_t accum_dis_tmp[val_per_thread] = {0};
        uint64_t accum_ref_dis_tmp[val_per_thread] = {0};

        uint32_t accum_ref_rd[val_per_thread / 2] = {0};
        uint32_t accum_dis_rd[val_per_thread / 2] = {0};

        const int stride_tmp = buf.stride_tmp / sizeof(uint32_t);
#pragma unroll
        for (int fj = 0; fj < fwidth_0; ++fj) {
#pragma unroll
            for (int off = 0; off < val_per_thread; ++off) {
                const int j = x_start + off;
                if (j < w) {
                    int jj = j - fwidth_0 / 2;
                    int jj_check = abs(jj + fj);
                    if (jj_check >= w) {
                        jj_check = 2 * w - jj_check - 2;
                    }
                    const uint16_t fcoeff = vif_filt_s0.filter[0][fj];
                    accum_mu1[off] +=
                        fcoeff * ((uint32_t)buf.tmp.mu1[y * stride_tmp + jj_check]);
                    accum_mu2[off] +=
                        fcoeff * ((uint32_t)buf.tmp.mu2[y * stride_tmp + jj_check]);
                    accum_ref_tmp[off] +=
                        fcoeff * ((uint64_t)buf.tmp.ref[y * stride_tmp + jj_check]);
                    accum_dis_tmp[off] +=
                        fcoeff * ((uint64_t)buf.tmp.dis[y * stride_tmp + jj_check]);
                    accum_ref_dis_tmp[off] +=
                        fcoeff * ((uint64_t)buf.tmp.ref_dis[y * stride_tmp + jj_check]);

                    if (fj >= (fwidth_0 - fwidth_1) / 2 &&
                            fj < (fwidth_0 - (fwidth_0 - fwidth_1) / 2) && off % 2 == 0) {
                        const uint16_t fcoeff_rd =
                            vif_filt_s0.filter[1][fj - ((fwidth_0 - fwidth_1) / 2)];
                        accum_ref_rd[off / 2] +=
                            fcoeff_rd * buf.tmp.ref_convol[y * stride_tmp + jj_check];
                        accum_dis_rd[off / 2] +=
                            fcoeff_rd * buf.tmp.dis_convol[y * stride_tmp + jj_check];
                    }
                }
            }
        }
        for (int off = 0; off < val_per_thread; ++off) {
            const int x = x_start + off;
            if (x < w) {
                accum_ref[off] = (uint32_t)((accum_ref_tmp[off] + 32768) >> 16);
                accum_dis[off] = (uint32_t)((accum_dis_tmp[off] + 32768) >> 16);
                accum_ref_dis[off] = (uint32_t)((accum_ref_dis_tmp[off] + 32768) >> 16);
                vif_statistic_calculation<uint32_t>(
                        accum_mu1[off], accum_mu2[off], accum_ref[off], accum_dis[off],
                        accum_ref_dis[off], x, w, h, vif_enhn_gain_limit, thread_accum);
            }
        }

        // reduce sums for each warp
        for (int i = 0; i < 7; ++i) {
            thread_accum_i64[i] = warp_reduce(thread_accum_i64[i]);
        }
        const int warp_id = threadIdx.x % VMAF_CUDA_THREADS_PER_WARP;
        // each warp writes its sum to global mem
        if (warp_id == 0) {
            for (int i = 0; i < 7; ++i) {
                atomicAdd_int64(&reinterpret_cast<int64_t *>(accum)[i],
                        thread_accum_i64[i]);
            }
        }

        uint16_t *ref = (uint16_t *)buf.ref;
        uint16_t *dis = (uint16_t *)buf.dis;
        for (int off = 0; off < val_per_thread; ++off) {
            const int x = x_start + off;
            if (y < h && x < w) {
                if ((y % 2) == 0 && (off % 2) == 0) {
                    const ptrdiff_t rd_stride = buf.stride / sizeof(uint16_t);
                    ref[(y / 2) * rd_stride + (x / 2)] =
                        (uint16_t)((accum_ref_rd[off / 2] + 32768) >> 16);
                    dis[(y / 2) * rd_stride + (x / 2)] =
                        (uint16_t)((accum_dis_rd[off / 2] + 32768) >> 16);
                }
            }
        }
    }
}

template <typename alignment_type = uint2, int fwidth, int fwidth_rd, int scale>
__device__ __forceinline__ void
filter1d_16_vertical_kernel(VifBufferCuda buf, uint16_t* ref_in, uint16_t* dis_in, int w, int h,
        int32_t add_shift_round_VP, int32_t shift_VP,
        int32_t add_shift_round_VP_sq, int32_t shift_VP_sq,
        filter_table_stuct vif_filt) {
    using writeback_type = uint4;
    constexpr int val_per_thread = sizeof(alignment_type) / sizeof(uint16_t);
    static_assert(val_per_thread % 4 == 0 && val_per_thread <= 8,
            "val per thread bust be divisible by 4 and under 16");
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int x_start = (blockIdx.x * blockDim.x + threadIdx.x) * val_per_thread;
    if (x_start < w && y < h) {
        __align__(sizeof(writeback_type)) uint32_t accum_mu1[val_per_thread] = {0};
        __align__(sizeof(writeback_type)) uint32_t accum_mu2[val_per_thread] = {0};
        __align__(sizeof(writeback_type)) uint32_t accum_ref_rd[val_per_thread] = {0};
        __align__(sizeof(writeback_type)) uint32_t accum_dis_rd[val_per_thread] = {0};
        uint64_t accum_ref[val_per_thread] = {0};
        uint64_t accum_dis[val_per_thread] = {0};
        uint64_t accum_ref_dis[val_per_thread] = {0};
        union {
            uint16_t ref[val_per_thread];
            alignment_type ref_aligned;
        };
        union {
            uint16_t dis[val_per_thread];
            alignment_type dis_aligned;
        };
        const ptrdiff_t stride = buf.stride / sizeof(uint16_t);
        for (int fi = 0; fi < fwidth; ++fi) {
            int ii = y - fwidth / 2;
            int ii_check = abs(ii + fi);
            if (ii_check >= h) {
                ii_check = 2 * h - ii_check - 2;
            }
            ref_aligned = *reinterpret_cast<alignment_type *>(
                    &(ref_in)[ii_check * stride + x_start]);
            dis_aligned = *reinterpret_cast<alignment_type *>(
                    &(dis_in)[ii_check * stride + x_start]);
            for (int off = 0; off < val_per_thread; ++off) {
                const int j = x_start + off;
                if (j < w) {
                    const uint16_t fcoeff = vif_filt.filter[scale][fi];
                    uint32_t imgcoeff_ref = ref[off];
                    uint32_t imgcoeff_dis = dis[off];
                    uint32_t img_coeff_ref = fcoeff * (uint32_t)imgcoeff_ref;
                    uint32_t img_coeff_dis = fcoeff * (uint32_t)imgcoeff_dis;
                    accum_mu1[off] += img_coeff_ref;
                    accum_mu2[off] += img_coeff_dis;
                    accum_ref[off] += img_coeff_ref * (uint64_t)imgcoeff_ref;
                    accum_dis[off] += img_coeff_dis * (uint64_t)imgcoeff_dis;
                    accum_ref_dis[off] += img_coeff_ref * (uint64_t)imgcoeff_dis;
                    if (fi >= (fwidth - fwidth_rd) / 2 &&
                            fi < (fwidth - (fwidth_rd - fwidth_rd) / 2) && fwidth_rd > 0) {
                        const uint16_t fcoeff_rd =
                            vif_filt.filter[scale + 1][fi - ((fwidth - fwidth_rd) / 2)];
                        accum_ref_rd[off] += fcoeff_rd * imgcoeff_ref;
                        accum_dis_rd[off] += fcoeff_rd * imgcoeff_dis;
                    }
                }
            }
        }
        const int stride_tmp = buf.stride_tmp / sizeof(uint32_t);

        __align__(sizeof(writeback_type)) uint32_t accum_ref_32[val_per_thread] = {0};
        __align__(sizeof(writeback_type)) uint32_t accum_dis_32[val_per_thread] = {0};
        __align__(sizeof(writeback_type)) uint32_t accum_ref_dis_32[val_per_thread] = {0};
        for (int off = 0; off < val_per_thread; ++off) {
            accum_mu1[off] =
                (uint16_t)((accum_mu1[off] + add_shift_round_VP) >> shift_VP);
            accum_mu2[off] =
                (uint16_t)((accum_mu2[off] + add_shift_round_VP) >> shift_VP);
            accum_ref_32[off] =
                (uint32_t)((accum_ref[off] + add_shift_round_VP_sq) >> shift_VP_sq);
            accum_dis_32[off] =
                (uint32_t)((accum_dis[off] + add_shift_round_VP_sq) >> shift_VP_sq);
            accum_ref_dis_32[off] =
                (uint32_t)((accum_ref_dis[off] + add_shift_round_VP_sq) >>
                        shift_VP_sq);
            if (fwidth_rd > 0) {
                accum_ref_rd[off] =
                    (uint16_t)((accum_ref_rd[off] + add_shift_round_VP) >> shift_VP);
                accum_dis_rd[off] =
                    (uint16_t)((accum_dis_rd[off] + add_shift_round_VP) >> shift_VP);
            }
        }

        for (int idx = 0; idx < val_per_thread; idx += sizeof(writeback_type) / sizeof(uint32_t)) {
            const int buffer_idx = y * stride_tmp + x_start + idx;
            if (x_start + idx < w) {
                *reinterpret_cast<writeback_type *>(&buf.tmp.mu1[buffer_idx]) =
                    *reinterpret_cast<writeback_type *>(&accum_mu1[idx]);
                *reinterpret_cast<writeback_type *>(&buf.tmp.mu2[buffer_idx]) =
                    *reinterpret_cast<writeback_type *>(&accum_mu2[idx]);
                *reinterpret_cast<writeback_type *>(&buf.tmp.ref[buffer_idx]) =
                    *reinterpret_cast<writeback_type *>(&accum_ref_32[idx]);
                *reinterpret_cast<writeback_type *>(&buf.tmp.dis[buffer_idx]) =
                    *reinterpret_cast<writeback_type *>(&accum_dis_32[idx]);
                *reinterpret_cast<writeback_type *>(&buf.tmp.ref_dis[buffer_idx]) =
                    *reinterpret_cast<writeback_type *>(&accum_ref_dis_32[idx]);
                if (fwidth_rd > 0) {
                    *reinterpret_cast<writeback_type *>(&buf.tmp.ref_convol[buffer_idx]) =
                        *reinterpret_cast<writeback_type *>(&accum_ref_rd[idx]);
                    *reinterpret_cast<writeback_type *>(&buf.tmp.dis_convol[buffer_idx]) =
                        *reinterpret_cast<writeback_type *>(&accum_dis_rd[idx]);
                }
            }
        }
    }
}

template <int val_per_thread = 2, int fwidth, int fwidth_rd, int scale>
__device__ __forceinline__ void
filter1d_16_horizontal_kernel(VifBufferCuda buf, int w, int h,
        int32_t add_shift_round_HP, int32_t shift_HP,
        filter_table_stuct vif_filt,
        double vif_enhn_gain_limit, vif_accums *accum) {
    static_assert(val_per_thread % 2 == 0,
            "val_per_thread must be divisible by 2");

    int y = blockIdx.y;
    int x_start = (blockIdx.x * blockDim.x + threadIdx.x) * val_per_thread;
    if (x_start < w && y < h) {
        union {
            vif_accums thread_accum;
            int64_t thread_accum_i64[7] = {0};
        };

        uint32_t accum_mu1[val_per_thread] = {0};
        uint32_t accum_mu2[val_per_thread] = {0};
        uint32_t accum_ref[val_per_thread] = {0};
        uint32_t accum_dis[val_per_thread] = {0};
        uint32_t accum_ref_dis[val_per_thread] = {0};
        uint64_t accum_ref_tmp[val_per_thread] = {0};
        uint64_t accum_dis_tmp[val_per_thread] = {0};
        uint64_t accum_ref_dis_tmp[val_per_thread] = {0};

        uint32_t accum_ref_rd[val_per_thread / 2] = {0};
        uint32_t accum_dis_rd[val_per_thread / 2] = {0};

        const int stride_tmp = buf.stride_tmp / sizeof(uint32_t);
#pragma unroll
        for (int fj = 0; fj < fwidth; ++fj) {
#pragma unroll
            for (int off = 0; off < val_per_thread; ++off) {
                const int j = x_start + off;
                if (j < w) {
                    int jj = j - fwidth / 2;
                    int jj_check = abs(jj + fj);
                    if (jj_check >= w) {
                        jj_check = 2 * w - jj_check - 2;
                    }
                    const uint16_t fcoeff = vif_filt.filter[scale][fj];
                    accum_mu1[off] +=
                        fcoeff * ((uint32_t)buf.tmp.mu1[y * stride_tmp + jj_check]);
                    accum_mu2[off] +=
                        fcoeff * ((uint32_t)buf.tmp.mu2[y * stride_tmp + jj_check]);
                    accum_ref_tmp[off] +=
                        fcoeff * ((uint64_t)buf.tmp.ref[y * stride_tmp + jj_check]);
                    accum_dis_tmp[off] +=
                        fcoeff * ((uint64_t)buf.tmp.dis[y * stride_tmp + jj_check]);
                    accum_ref_dis_tmp[off] +=
                        fcoeff * ((uint64_t)buf.tmp.ref_dis[y * stride_tmp + jj_check]);

                    if (fj >= (fwidth - fwidth_rd) / 2 &&
                            fj < (fwidth - (fwidth - fwidth_rd) / 2) && fwidth_rd > 0 &&
                            off % 2 == 0) {
                        const uint32_t fcoeff_rd =
                            vif_filt.filter[scale + 1][fj - ((fwidth - fwidth_rd) / 2)];
                        accum_ref_rd[off / 2] +=
                            fcoeff_rd *
                            ((uint32_t)buf.tmp.ref_convol[y * stride_tmp + jj_check]);
                        accum_dis_rd[off / 2] +=
                            fcoeff_rd *
                            ((uint32_t)buf.tmp.dis_convol[y * stride_tmp + jj_check]);
                    }
                }
            }
        }
        for (int off = 0; off < val_per_thread; ++off) {
            const int x = x_start + off;
            if (x < w) {
                accum_ref[off] =
                    (uint32_t)((accum_ref_tmp[off] + add_shift_round_HP) >> shift_HP);
                accum_dis[off] =
                    (uint32_t)((accum_dis_tmp[off] + add_shift_round_HP) >> shift_HP);
                accum_ref_dis[off] =
                    (uint32_t)((accum_ref_dis_tmp[off] + add_shift_round_HP) >>
                            shift_HP);
                vif_statistic_calculation<uint32_t>(
                        accum_mu1[off], accum_mu2[off], accum_ref[off], accum_dis[off],
                        accum_ref_dis[off], x, w, h, vif_enhn_gain_limit, thread_accum);
            }
        }

        // reduce sums for each warp
        for (int i = 0; i < 7; ++i) {
            thread_accum_i64[i] = warp_reduce(thread_accum_i64[i]);
        }
        const int warp_id = threadIdx.x % VMAF_CUDA_THREADS_PER_WARP;
        // each warp writes its sum to global mem
        if (warp_id == 0) {
            for (int i = 0; i < 7; ++i) {
                atomicAdd_int64(&reinterpret_cast<int64_t *>(accum)[i],
                        thread_accum_i64[i]);
            }
        }

        for (int off = 0; off < val_per_thread; ++off) {
            const int x = x_start + off;
            if (y < h && x < w) {
                if ((y % 2) == 0 && (off % 2) == 0) {
                    uint16_t *ref = (uint16_t *)buf.ref;
                    uint16_t *dis = (uint16_t *)buf.dis;
                    const ptrdiff_t rd_stride = buf.stride / sizeof(uint16_t);
                    ref[(y / 2) * rd_stride + (x / 2)] =
                        (uint16_t)((accum_ref_rd[off / 2] + 32768) >> 16);
                    dis[(y / 2) * rd_stride + (x / 2)] =
                        (uint16_t)((accum_dis_rd[off / 2] + 32768) >> 16);
                }
            }
        }
    }
}

#define FILTER1D_8_VERT(alignment_type, fwidth_0, fwidth_1)                                 \
    __global__ void filter1d_8_vertical_kernel_##alignment_type##_##fwidth_0##_##fwidth_1 ( \
            VifBufferCuda buf, uint8_t* ref_in, uint8_t* dis_in,                            \
            int w, int h, filter_table_stuct vif_filt_s0)                                   \
{                                                                                           \
    filter1d_8_vertical_kernel<alignment_type, fwidth_0, fwidth_1>(                         \
            buf, ref_in, dis_in, w, h, vif_filt_s0);                                        \
}

#define FILTER1D_8_HORI(val_per_thread, fwidth_0, fwidth_1)                                   \
    __global__ void filter1d_8_horizontal_kernel_##val_per_thread##_##fwidth_0##_##fwidth_1 ( \
            VifBufferCuda buf, int w, int h,  filter_table_stuct vif_filt_s0,                 \
            double vif_enhn_gain_limit,  vif_accums *accum)                                   \
{                                                                                             \
    filter1d_8_horizontal_kernel<val_per_thread, fwidth_0, fwidth_1>(                         \
            buf, w, h, vif_filt_s0, vif_enhn_gain_limit, accum);                              \
}

#define FILTER1D_16_VERT(alignment_type, fwidth, fwidth_rd, scale)                                    \
    __global__ void filter1d_16_vertical_kernel_##alignment_type##_##fwidth##_##fwidth_rd##_##scale ( \
            VifBufferCuda buf, uint16_t* ref_in, uint16_t* dis_in, int w, int h,                      \
            int32_t add_shift_round_VP, int32_t shift_VP,                                             \
            int32_t add_shift_round_VP_sq, int32_t shift_VP_sq,                                       \
            filter_table_stuct vif_filt)                                                              \
{                                                                                                     \
    filter1d_16_vertical_kernel<alignment_type, fwidth, fwidth_rd, scale>(                            \
            buf, ref_in, dis_in, w, h, add_shift_round_VP, shift_VP,                                  \
            add_shift_round_VP_sq, shift_VP_sq, vif_filt);                                            \
}

#define FILTER1D_16_HORI(val_per_thread, fwidth, fwidth_rd, scale)                                      \
    __global__ void filter1d_16_horizontal_kernel_##val_per_thread##_##fwidth##_##fwidth_rd##_##scale ( \
            VifBufferCuda buf, int w, int h,                                                            \
            int32_t add_shift_round_HP, int32_t shift_HP,                                               \
            filter_table_stuct vif_filt,                                                                \
            double vif_enhn_gain_limit, vif_accums *accum)                                              \
{                                                                                                       \
    filter1d_16_horizontal_kernel<val_per_thread, fwidth, fwidth_rd, scale>(                            \
            buf, w, h, add_shift_round_HP, shift_HP,                                                    \
            vif_filt, vif_enhn_gain_limit, accum);                                                      \
}

extern "C" {
    // constexpr int fwidth[4] = {17, 9, 5, 3};
    FILTER1D_8_VERT(uint32_t, 17, 9);   // filter1d_8_vertical_kernel_uint32_t_17_9
    FILTER1D_8_HORI(2, 17, 9);          // filter1d_8_horizontal_kernel_2_17_9
    FILTER1D_16_VERT(uint2, 17, 9, 0);  // filter1d_16_vertical_kernel_uint2_17_9_0
    FILTER1D_16_VERT(uint2, 9, 5, 1);   // filter1d_16_vertical_kernel_uint2_9_5_1
    FILTER1D_16_VERT(uint2, 5, 3, 2);   // filter1d_16_vertical_kernel_uint2_5_3_2
    FILTER1D_16_VERT(uint2, 3, 0, 3);   // filter1d_16_vertical_kernel_uint2_3_0_3

    FILTER1D_16_HORI(2, 17, 9, 0);      // filter1d_16_horizontal_kernel_2_17_9_0
    FILTER1D_16_HORI(2, 9, 5, 1);       // filter1d_16_horizontal_kernel_2_9_5_1
    FILTER1D_16_HORI(2, 5, 3, 2);       // filter1d_16_horizontal_kernel_2_5_3_2
    FILTER1D_16_HORI(2, 3, 0, 3);       // filter1d_16_horizontal_kernel_2_3_0_3
}
